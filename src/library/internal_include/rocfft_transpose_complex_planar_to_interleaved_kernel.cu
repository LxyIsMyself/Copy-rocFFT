#include "hip/hip_runtime.h"

//works for complex planar to complex interleaved; T should be float or double; T2 should be float2 or double2
template<typename T, typename T2, int micro_tile_col_size, int micro_tile_row_size, int wg_col_size, int wg_row_size>
__global__ void transpose_kernel_outplace_complex_planar_to_complex_interleaved(hipLaunchParm lp,
                                                                                T *input_matrix_real,
	            								T *input_matrix_imag,
									        T2 *output_matrix,
										size_t input_row_size,
										size_t input_col_size,
										size_t input_leading_dim_size,
										size_t output_leading_dim_size,
										size_t batch_size)
{
    // WG size can be assumed to be 16 by 16 in most cases
    size_t local_idx_0 = hipThreadIdx_x;// 0-15
    size_t local_idx_1 = hipThreadIdx_y;// 0-15
    size_t block_idx_0 = hipBlockIdx_x;// index of work groups
    size_t block_idx_1 = hipBlockIdx_y;
    size_t block_dim_0 = hipBlockDim_x;// size of work groups 16
    size_t block_dim_1 = hipBlockDim_y;// size of work groups 16
    size_t grid_dim_0 = hipGridDim_x;// number of blocks only the first dimension is loaded to handle batch size

    // for 64 x 64 macro tile size we will need 16 x 4 x 64 blocks (4 x 64 == 16 x 16)
    // for 32 x 32 macro tile size we will need  4 x 8 x 32 blocks
    const size_t macro_tile_col_size = micro_tile_col_size * wg_col_size;
    const size_t macro_tile_row_size = micro_tile_row_size * wg_row_size;
    const size_t reshape_factor = macro_tile_col_size / block_dim_0; // 64 / 16 = 4 need to fit 4 rows into one row in LDS; 32 / 16 = 2
    const size_t unroll_factor = macro_tile_row_size / (block_dim_1 / reshape_factor); // 64 / (16 / 4) = 16; 32 / (16 / 2) = 4

	__shared__ T lds_real[macro_tile_row_size][macro_tile_col_size];
	__shared__ T lds_imag[macro_tile_row_size][macro_tile_col_size];

	size_t blocks_per_batch = grid_dim_0 / batch_size;
	size_t batch_idx = block_idx_0 / blocks_per_batch;

	input_matrix_real += batch_idx * input_leading_dim_size * input_row_size;
	input_matrix_imag += batch_idx * input_leading_dim_size * input_row_size;

	size_t input_offset = 0;
	input_offset += input_leading_dim_size * block_idx_1 * macro_tile_row_size;
	input_offset += (block_idx_0 % blocks_per_batch) * macro_tile_col_size;

	input_matrix_real += input_offset;
	input_matrix_imag += input_offset;

	for(int i = 0; i < unroll_factor; i++)
	{
	        //each iteration 256 work items will read from a 4 x 64 subblock
                //there are 16 iterations
		size_t subblock_idx_0 = local_idx_0 + (local_idx_1 % reshape_factor) * block_dim_0; // local_idx_0 + (local_idx_1 % 4) * 16
                size_t subblock_idx_1 = local_idx_1 / reshape_factor + i * (block_dim_1 / reshape_factor);
		//transpose happened here
		lds_real[subblock_idx_0][subblock_idx_1] = input_matrix_real[subblock_idx_1 * input_leading_dim_size + subblock_idx_0];
		lds_imag[subblock_idx_0][subblock_idx_1] = input_matrix_imag[subblock_idx_1 * input_leading_dim_size + subblock_idx_0];
	}

	__syncthreads();

	output_matrix += batch_idx * input_col_size * output_leading_dim_size;
	size_t output_offset = 0;
        output_offset += output_leading_dim_size * (block_idx_0 % blocks_per_batch) * macro_tile_row_size;//input_row_size == ouput_col_size
        output_offset += block_idx_1 * macro_tile_col_size;

	output_matrix += output_offset;

	for(int i = 0; i < unroll_factor; i++)
	{
	    size_t subblock_idx_0 = local_idx_0 + (local_idx_1 % reshape_factor) * block_dim_0;// 0-63
            size_t subblock_idx_1 = local_idx_1 / reshape_factor + i * (block_dim_1 / reshape_factor);// 0-3, 4-7 ... 60-63
	    output_matrix[subblock_idx_1 * output_leading_dim_size + subblock_idx_0].x = lds_real[subblock_idx_1][subblock_idx_0];
	    output_matrix[subblock_idx_1 * output_leading_dim_size + subblock_idx_0].y = lds_imag[subblock_idx_1][subblock_idx_0];
	}
}
