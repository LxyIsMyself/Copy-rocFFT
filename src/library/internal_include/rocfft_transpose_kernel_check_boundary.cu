#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

template<typename T, int micro_tile_col_size, int micro_tile_row_size, int wg_col_size, int wg_row_size>
__global__ void transpose_kernel_outplace_check_boundary(hipLaunchParm lp, T *input_matrix, T *output_matrix, size_t input_row_size, size_t input_col_size, size_t input_leading_dim_size, size_t output_leading_dim_size, size_t batch_size)
{
//TODO
}
